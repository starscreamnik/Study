#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <algorithm>
#include <hip/hip_runtime.h>

#include "hip/hip_runtime.h"
#include ""

using namespace std;

const int N = 33 * 1024;
const int threadsPerBlock = 256;
const int blockPerGrid = min(32, (N + threadsPerBlock - 1) / threadsPerBlock);

void errorHandler(hipError_t error, const string& msg) {
	if (error != hipSuccess) {
		cerr << msg << endl;
		exit(0);
	}
}

void input(ifstream& in, const int dim, int*& m1, int*& m2, int*& mAns, int*& v, int*& vAns) {
	for (int i = 0; i < dim*dim; i++) in >> m1[i];
	for (int i = 0; i < dim*dim; i++) in >> m2[i];
	for (int i = 0; i < dim; i++) in >> v[i];
	memset(vAns, 0, dim * sizeof(int));
	memset(mAns, 0, dim*dim * sizeof(int));
}

void outputMxM(const int dim, int *a, int *b, int *ans) {
	cout << "============Matrix on Matrix=============" << endl;
	cout << "Parallel:\n";
	for (int i = 0; i < dim; i++) {
		for (int j = 0; j < dim; j++) {
			cout << ans[i*dim + j] << " ";
		}
		cout << endl;
	}

	cout << "Consistent:\n";
	for (int i = 0; i < dim; i++) {
		for (int j = 0; j < dim; j++) {
			int c = 0;
			for (int k = 0; k < dim; k++) {
				c += a[dim*i + k] * b[dim*k + j];
			}
			cout << c << " ";
		}
		cout << "\n";
	}
	cout << endl;
}

void outputMxV(const int dim, int *m, int *v, int *ans) {
	cout << "============Matrix on Vector=============" << endl;
	cout << "Parallel:\n";
	for (int i = 0; i < dim; i++) {
		cout << ans[i] << " ";
	}
	cout << endl;

	cout << "Consistent:\n";
	for (int i = 0; i < dim; i++) {
		int c = 0;
		for (int j = 0; j < dim; j++) {
			c += m[i*dim + j] * v[j];
		}
		cout << c << " ";
	}
	cout << endl;
}

__device__ int findRow(int tid, const int n) {
	int counter = 0;
	while (tid >= n) {
		tid -= n;
		counter++;
	}
	return counter;
}

__global__ void MxMKernel(const int *m1, const int *m2, int *c, int n) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	int i = tid, j = tid;
	if (tid < n*n) {
		if (tid >= n) {
			j %= n;
			i = findRow(tid, n);
		}
		else i = 0;

		for (int k = 0; k < n; k++)
			c[tid] += m1[i*n + k] * m2[k*n + j];
	}
}

__global__ void MxVKernel(const int *m, const int *v, int *ans, int n) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < n) {
		for (int k = 0; k < n; k++)
			ans[tid] += m[tid*n + k] * v[k];
	}
}

int main() {
	ifstream in("input.txt");
	int dim;
	in >> dim;
	const size_t dSize = sizeof(int)*dim*dim;
	int *m1 = (int*)malloc(dSize), *m2 = (int*)malloc(dSize), *mAns = (int*)malloc(dSize);
	int *v = (int*)malloc(sizeof(int)*dim), *vAns = (int*)malloc(sizeof(int)*dim);

	input(in, dim, m1, m2, mAns, v, vAns);

	int *cuM1 = nullptr, *cuM2 = nullptr, *cuMAns = nullptr, *cuV = nullptr, *cuVAns = nullptr;

	errorHandler(hipMalloc(&cuM1, dSize), "hipMalloc: cuM1");
	errorHandler(hipMalloc(&cuM2, dSize), "hipMalloc: cuM2");
	errorHandler(hipMalloc(&cuMAns, dSize), "hipMalloc: cuMAns");
	errorHandler(hipMalloc(&cuV, sizeof(int)*dim), "hipMalloc: cuV");
	errorHandler(hipMalloc(&cuVAns, sizeof(int)*dim), "hipMalloc: cuVAns");

	errorHandler(hipMemcpy(cuM1, m1, dSize, hipMemcpyHostToDevice), "hipMemcpy: toDevice: cuM1");
	errorHandler(hipMemcpy(cuM2, m2, dSize, hipMemcpyHostToDevice), "cudaMemCpy: toDevice: cuM2");
	errorHandler(hipMemcpy(cuMAns, mAns, dSize, hipMemcpyHostToDevice), "cudaMemCpy: toDevice: cuMAns");
	errorHandler(hipMemcpy(cuV, v, sizeof(int)*dim, hipMemcpyHostToDevice), "hipMemcpy: toDevice: cuV");
	errorHandler(hipMemcpy(cuVAns, vAns, sizeof(int)*dim, hipMemcpyHostToDevice), "hipMemcpy: toDevice: cuVAns");

	MxMKernel << <blockPerGrid, threadsPerBlock >> > (cuM1, cuM2, cuMAns, dim);
	errorHandler(hipGetLastError(), "hipGetLastError");
	errorHandler(hipMemcpy(mAns, cuMAns, dSize, hipMemcpyDeviceToHost), "hipMemcpy: toHost: mAns");
	outputMxM(dim, m1, m2, mAns);

	MxVKernel << <blockPerGrid, threadsPerBlock >> > (cuM1, cuV, cuVAns, dim);
	errorHandler(hipGetLastError(), "hipGetLastError");
	errorHandler(hipMemcpy(vAns, cuVAns, sizeof(int)*dim, hipMemcpyDeviceToHost), "hipMemcpy: toHost: vAns");
	outputMxV(dim, m1, v, vAns);

	free(m1), free(m2), free(mAns), free(v), free(vAns);
	errorHandler(hipFree(cuM1), "hipFree: cuM1");
	errorHandler(hipFree(cuM2), "hipFree: cuM2");
	errorHandler(hipFree(cuMAns), "hipFree: cuMAns");
	errorHandler(hipFree(cuV), "hipFree: cuV");
	errorHandler(hipFree(cuVAns), "hipFree: cuVAns");

	return 0;
}
